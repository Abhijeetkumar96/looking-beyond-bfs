#include "hip/hip_runtime.h"
#include "reversePaths.h"
#include "../util/utility.h"

__global__
void Reverse(int* onPath, int *parent,int *new_parent,int n) {

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < n) {
		if(onPath[tid])
		{
			if(parent[tid] != tid)
				new_parent[parent[tid]] = tid;
		}
	}
}

__global__
void MarkOnPath(int* onPath,int* newOnPath, int* pr_arr,int n, int log_n,int *iter_no) {

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < n) {
		if(onPath[tid]) {
            long long j = (log_n * 1LL * tid) + iter_no[tid];
			if((j < log_n * 1LL * (tid + 1)) && (j >= log_n * 1LL * tid))
			{
				if(pr_arr[j]!=-1)
				{
					newOnPath[pr_arr[j]] = 1;
					iter_no[pr_arr[j]] = iter_no[tid];
				}
			}
        }
	}
}

__global__ 
void DecrementIter(int n,int * onPath,int *iter_no)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < n) {
		iter_no[tid] = max(0,iter_no[tid]-1);
	}
}

void ReversePaths(
	int vertices,
	int edges,
	int log_2_size,
	int *d_OnPath,
	int *d_new_OnPath,
	int *d_pr_arr,
	int *d_parent_ptr,
	int *d_new_parent_ptr,
	int *d_index_ptr,
	int *d_pr_size_ptr
)
{
		#ifdef DEBUG
			std::vector<int> onPath(vertices),rnodes,prnodes,pr_arr(log_2_size*vertices),pr_size(vertices);
		#endif

		int numThreads = 1024;
		int numBlocks_n = (vertices + numThreads - 1) / numThreads;
		// int numBlocks_e = (edges + numThreads - 1) / numThreads;

		hipMemcpy(d_new_OnPath,d_OnPath, sizeof(int) * vertices, hipMemcpyDeviceToDevice);

		#ifdef DEBUG
			hipMemcpy(onPath.data(), d_OnPath, vertices*sizeof(int), hipMemcpyDeviceToHost);
			std::cout << "OnPath nodes intially : \n";
			for(int i=0;i< vertices;i++)
			{
				if(onPath[i] == 1)
				{
					rnodes.push_back(i);
				}
			}
			printArr(rnodes,rnodes.size(),10);
		#endif		

		// Step 3.1: Mark OnPath array, OnPath[u...v] = {1,...,1} denotes path from u to v needs to be reversed
		for (int j = 0; j < log_2_size ; ++j) {
			
			DecrementIter<<<numBlocks_n, numThreads>>> (vertices ,d_OnPath, d_pr_size_ptr);
			hipDeviceSynchronize();
			
			MarkOnPath<<<numBlocks_n, numThreads>>> (d_OnPath,d_new_OnPath, d_pr_arr, vertices, log_2_size,d_pr_size_ptr);
			hipDeviceSynchronize();
			
			#ifdef DEBUG
				hipMemcpy(onPath.data(),d_new_OnPath, sizeof(int) * vertices, hipMemcpyDeviceToHost);
				std::cout<<"Iteration Number : "<<j<<"\n";
				printArr(onPath,vertices,10);
			#endif

			hipMemcpy(d_OnPath,d_new_OnPath, sizeof(int) * vertices, hipMemcpyDeviceToDevice);

		}
		// hipMemcpy(d_new_OnPath,d_OnPath, sizeof(int) * vertices, hipMemcpyDeviceToDevice);

		#ifdef DEBUG
			hipMemcpy(onPath.data(), d_OnPath, vertices*sizeof(int), hipMemcpyDeviceToHost);
			std::cout << "OnPath nodes after : \n";
			rnodes.clear();
			// print(h_onPath_arr);
			for(int i=0;i < vertices;i++)
			{
				if(onPath[i] == 1)
				{
					rnodes.push_back(i);
				}
			}
			printArr(rnodes,rnodes.size(),10);
		#endif
		// // Step 3.2: Reverse the marked paths

		hipMemcpy(d_new_parent_ptr,d_parent_ptr, sizeof(int) * vertices, hipMemcpyDeviceToDevice);
		
		Reverse<<<numBlocks_n, numThreads>>> (d_OnPath,d_parent_ptr, d_new_parent_ptr,vertices);
		hipDeviceSynchronize();

		hipMemcpy(d_parent_ptr,d_new_parent_ptr, sizeof(int) * vertices, hipMemcpyDeviceToDevice);
}
