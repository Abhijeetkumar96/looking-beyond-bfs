#include "hip/hip_runtime.h"
#include "rootedSpanningTreePR.h"
#include "grafting.h"
#include "reRoot.h"
#include "shortcutting.h"
#include "../util/utility.h"
#include "../verification/verifySol.h"

__global__ void init(int *arr, int *rep, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n)
	{
		arr[tid] = tid;
		rep[tid] = tid;
	}
}

std::vector<int> RootedSpanningTree(const std::vector<int> &u_arr, const std::vector<int> &v_arr, const int n)
{

	int vertices = n;
	int edges = u_arr.size();

	thrust::device_vector<int> d_u_arr(edges);
	thrust::device_vector<int> d_v_arr(edges);
	thrust::device_vector<int> d_winner(n, 0);

	thrust::copy(u_arr.begin(),u_arr.end(),d_u_arr.begin());
	thrust::copy(v_arr.begin(),v_arr.end(),d_v_arr.begin());

	int *d_u_ptr = thrust::raw_pointer_cast(d_u_arr.data());
	int *d_v_ptr = thrust::raw_pointer_cast(d_v_arr.data());
	int *d_winner_ptr = thrust::raw_pointer_cast(d_winner.data());

	std::cout << "log2(n) = " << std::log2(n) << std::endl;
	int log_2_size = std::ceil(std::log2(n));
	long long pr_size = std::ceil(n * 1LL * log_2_size * sizeof(int));
	// std::cout << "pr_size = " << pr_size/sizeof(int) << std::endl;
	
	long long size = n * 1LL * sizeof(int);


	int *d_ptr;
	int *d_parent_ptr;
	int *d_new_parent_ptr;
	int *d_pr_arr;
	int *d_label;
	int *d_OnPath;
	int *d_new_OnPath;
	int *d_rep;
	int *d_marked_parent;
	int *d_next;
	int *d_new_next;
	int *d_index_ptr;
	int *d_pr_size_ptr;


	hipMalloc((void **)&d_ptr, size);
	hipMalloc((void **)&d_parent_ptr,size);
	hipMalloc((void **)&d_new_parent_ptr,size);
	hipMalloc((void **)&d_pr_arr, pr_size);
	hipMalloc((void **)&d_label, size);
	hipMalloc((void **)&d_rep, size);
	hipMalloc((void **)&d_OnPath, size);
	hipMalloc((void **)&d_new_OnPath, size);
	hipMalloc((void **)&d_marked_parent,size);
	hipMalloc((void **)&d_next, size);
	hipMalloc((void **)&d_new_next, size);
	hipMalloc((void **)&d_index_ptr, size);
	hipMalloc((void **)&d_pr_size_ptr, size);

#ifdef DEBUG
	std::vector<int> rep(n),par(n),marked(n),pr_arr(pr_size),pr_arr_size(n);
#endif

	int grafting_time = 0, shortcutting_time = 0 , reroot_time = 0;

	int numThreads = 1024;
	int numBlocks_n = (vertices + numThreads - 1) / numThreads;

	auto start = std::chrono::high_resolution_clock::now();

	// Step 1: Initialize rep with vertices themselves
	init<<<numBlocks_n, numThreads>>>(d_ptr, d_parent_ptr, vertices);
	hipDeviceSynchronize();


#ifdef DEBUG
	std::cout << "Rep array initially : \n";
	hipMemcpy(rep.data(), d_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);
	printArr(rep,vertices,10);
#endif

	int *d_flag;
	hipMalloc(&d_flag, sizeof(int));

	int flag = 1;
	int iter_number = 0, isMaxIteration = 0;
	// int numBlocks_e = (edges + numThreads - 1) / numThreads;

	while (flag)
	{
		if(iter_number > 2*log_2_size)
		{
			std::cout<<"Iterations exceeded 2*log_2_n : "<<iter_number<<"\n";
			break;
		}

		isMaxIteration ^= 1;

		#ifdef DEBUG
			std::cout<<"\nIteration : "<<iter_number<<"\n";
		#endif

		#ifdef DEBUG
			hipMemcpy(rep.data(), d_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);
			std::cout<<"No of components intially : "<<numberOfComponents(rep)<<"\n";
		#endif

		flag = 0;
		hipMemcpy(d_flag, &flag, sizeof(int), hipMemcpyHostToDevice);
		hipMemset(d_OnPath, 0, size);
		hipMemset(d_index_ptr,0,size);
		hipMemset(d_marked_parent,-1,size);
		
		//thrust::fill is better optimized than hipMemset
		thrust::fill(d_winner.begin(),d_winner.end(), -1);

		//Step 2: Graft

		auto start_graft = std::chrono::high_resolution_clock::now();

		Graft(vertices,edges,d_u_ptr,d_v_ptr,d_ptr,d_winner_ptr,d_marked_parent,d_OnPath,d_flag,isMaxIteration);
		
		auto end_graft = std::chrono::high_resolution_clock::now();
		auto duration_graft = std::chrono::duration_cast<std::chrono::milliseconds>(end_graft - start_graft).count();
	
		grafting_time += duration_graft;
	
		#ifdef DEBUG
			hipMemcpy(marked.data(), d_marked_parent, sizeof(int) * n, hipMemcpyDeviceToHost);
			std::cout<<"No of marked components : "<<markedComponents(marked)<<"\n";
		#endif
		
		#ifdef DEBUG
    		std::cout << "Marked parent array :\n";
			for(int i=0;i<n;i++)
			{
				if(marked[i] != -1)
					std::cout<< i << " : " << marked[i] << "\n";
			}
		#endif

		// Step 3: ReRoot
		auto start_reroot = std::chrono::high_resolution_clock::now();

		ReRoot(vertices,edges,log_2_size,iter_number,d_OnPath,d_new_OnPath ,d_pr_arr,d_parent_ptr,d_new_parent_ptr,d_index_ptr,d_pr_size_ptr,d_marked_parent,d_ptr);

		auto end_reroot = std::chrono::high_resolution_clock::now();
		auto duration_reroot = std::chrono::duration_cast<std::chrono::milliseconds>(end_reroot - start_reroot).count();
	
		reroot_time += duration_reroot;
		#ifdef DEBUG		
			hipMemcpy(par.data(), d_parent_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);		
			std::cout<<"No of roots after rerooting : "<<rootedComponents(par)<<"\n";
		#endif
		
		hipMemcpy(d_next, d_parent_ptr, size, hipMemcpyDeviceToDevice);

		#ifdef DEBUG
    		std::cout << "Parent array after rerooting : ";
			hipMemcpy(par.data(), d_parent_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);
			printArr(par,vertices,10);
		#endif

		#ifdef DEBUG
	    	std::cout <<"Rep array before shortcutting: ";
			hipMemcpy(rep.data(), d_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);
			printArr(rep,vertices,10);
	    #endif

		// Step 4: Shortcutting
		hipMemset(d_pr_size_ptr,0,size);
		hipMemset(d_pr_arr, -1, pr_size);

		auto start_shortcut = std::chrono::high_resolution_clock::now();
		
		Shortcut(vertices,edges,log_2_size,d_next,d_new_next,d_pr_arr,d_ptr,d_pr_size_ptr);	
		
		auto end_shortcut = std::chrono::high_resolution_clock::now();
		auto duration_shortcut = std::chrono::duration_cast<std::chrono::milliseconds>(end_shortcut - start_shortcut).count();
	
		shortcutting_time += duration_shortcut;

		#ifdef DEBUG
			hipMemcpy(rep.data(), d_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);		
			std::cout<<"No of roots after shortcutting: "<<numberOfComponents(rep)<<"\n";	
		#endif

		#ifdef DEBUG
	    	std::cout <<"Rep array after shortcutting: ";
			hipMemcpy(rep.data(), d_ptr, sizeof(int) * n, hipMemcpyDeviceToHost);
			printArr(rep,vertices,10);
	    #endif

		#ifdef DEBUG
	    	hipMemcpy(pr_arr.data(), d_pr_arr, sizeof(int) * pr_size, hipMemcpyDeviceToHost);
			hipMemcpy(pr_arr_size.data(), d_pr_size_ptr, size, hipMemcpyDeviceToHost);
	    	printPR(pr_arr,pr_arr_size,vertices,log_2_size);
	    #endif
		
		iter_number++;
		hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
		
		#ifdef DEBUG
			std::cout << "Flag = " << flag << std::endl;
		#endif
		// break;
	}
	
	auto end = std::chrono::high_resolution_clock::now();
	
	auto duration  = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::vector<int> h_parent(n),h_rep(n);
	hipMemcpy(h_parent.data(), d_parent_ptr, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_rep.data(), d_ptr, n*sizeof(int), hipMemcpyDeviceToHost);
	

#ifdef DEBUG
	std::cout << "parent array : \n";

	int j = 0;
	for (auto i : h_parent)
		std::cout << "parent[" << j++ << "] = " << i << std::endl;
	std::cout << std::endl;
#endif

#ifdef DEBUG
	std::cout << "rep array : \n";
	
	j = 0;
	for (auto i : h_rep)
		std::cout << "rep[" << j++ << "] = " << i << std::endl;
	std::cout << std::endl;
#endif
	std::cout << "Number of iterations taken: "<<iter_number <<"\n";
	std::cout << "Duration in milliseconds: " << duration << " ms\n";
	

	hipFree(d_OnPath);
	return h_parent;
}
