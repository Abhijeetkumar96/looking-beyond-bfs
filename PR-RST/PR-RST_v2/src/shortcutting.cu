#include "hip/hip_runtime.h"
#include "shortcutting.h"
#include "utility.h"

//Pointer Jumping Kernel
__global__
void UpdatePR(int *next,int *new_next, int* pr_arr, int log_n, int itr_no, int n,int *pr_size) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;    
    if(tid < n)
    {
        long long starting_index = log_n * 1LL * tid;
        long long next_index = log_n * 1LL * (tid+1);
		if((starting_index + itr_no) < next_index)
		{
			pr_arr[starting_index + itr_no] = next[tid];
		}
        if(next[tid] != tid)
        {
			new_next[tid] = next[next[tid]];
			if(next[tid] != next[next[tid]])
				pr_size[tid]++;
        }
		else
		{
			new_next[tid] = tid; 
		}
    }
}

void Shortcut(
	int vertices,
	int edges,
	int log_2_size,
	int *d_next,
	int *d_new_next,
	int *d_pr_arr,
	int *d_ptr,
	int *d_pr_size_ptr)
{
		int numThreads = 1024;
		int numBlocks_n = (vertices + numThreads - 1) / numThreads;
		// int numBlocks_e = (edges + numThreads - 1) / numThreads;
		
        // Step 4.1: Shortcut PR
	
		for (int j = 0; j < log_2_size; ++j) {
	        UpdatePR<<<numBlocks_n, numThreads>>> (d_next,d_new_next, d_pr_arr, log_2_size, j, vertices, d_pr_size_ptr);
	        CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize UpdatePR array");
			int * temp = d_new_next;
			d_new_next = d_next;
			d_next = temp;
		}

		// hipMemcpy(d_new_next,d_next, sizeof(int) * vertices, hipMemcpyDeviceToDevice);

		CUDA_CHECK(hipMemcpy(d_ptr,d_next, sizeof(int) * vertices, hipMemcpyDeviceToDevice),
			"Failed to copy d_ptr/d_next array");
}